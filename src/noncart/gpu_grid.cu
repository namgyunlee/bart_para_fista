#include "hip/hip_runtime.h"
/* Copyright 2022. Uecker Lab. University Medical Center Göttingen.
 * All rights reserved. Use of this source code is governed by
 * a BSD-style license which can be found in the LICENSE file.
 *
 * Authors: Moritz Blumenthal
 */

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

#include <assert.h>
#include <stdbool.h>

#include "misc/misc.h"

#include "num/multind.h"
#include "num/gpu_misc.h"
#include "num/gpuops.h"
#include "num/multiplace.h"

#include "noncart/grid.h"
#include "gpu_grid.h"

__device__ hipFloatComplex zexp(hipFloatComplex x)
{
	float sc = expf(hipCrealf(x));
	float si;
	float co;
	sincosf(hipCimagf(x), &si, &co);
	return make_hipFloatComplex(sc * co, sc * si);
}

struct linphase_conf {

	long dims[3];
	long tot;
	float shifts[3];
	long N;
	float cn;
	float scale;
	_Bool conj;
	_Bool fmac;
};

__global__ void kern_apply_linphases_3D(struct linphase_conf c, hipFloatComplex* dst, const hipFloatComplex* src)
{
	int startX = threadIdx.x + blockDim.x * blockIdx.x;
	int strideX = blockDim.x * gridDim.x;

	int startY = threadIdx.y + blockDim.y * blockIdx.y;
	int strideY = blockDim.y * gridDim.y;

	int startZ = threadIdx.z + blockDim.z * blockIdx.z;
	int strideZ = blockDim.z * gridDim.z;

	for (long z = startZ; z < c.dims[2]; z += strideZ)
		for (long y = startY; y < c.dims[1]; y += strideY)
			for (long x = startX; x < c.dims[0]; x +=strideX) {

				long pos[3] = { x, y, z };
				long idx = x + c.dims[0] * (y + c.dims[1] * z);
				
				float val = c.cn;

				for (int n = 0; n < 3; n++)
					val += pos[n] * c.shifts[n];

				if (c.conj)
					val = -val;
				
				hipFloatComplex cval = make_hipFloatComplex(0, val);
				cval = zexp(cval);

				cval.x *= c.scale;
				cval.y *= c.scale;

				if (c.fmac) {

					for (long i = 0; i < c.N; i++)
						dst[idx + i * c.tot] = hipCaddf(dst[idx + i * c.tot], hipCmulf(src[idx + i * c.tot], cval));
				} else {

					for (long i = 0; i < c.N; i++)
						dst[idx + i * c.tot] = hipCmulf(src[idx + i * c.tot], cval);
				}
			}
}



extern "C" void cuda_apply_linphases_3D(int N, const long img_dims[], const float shifts[3], _Complex float* dst, const _Complex float* src, _Bool conj, _Bool fmac, float scale)
{
	struct linphase_conf c;

	c.cn = 0;
	c.tot = 1;
	c.N = 1;
	c.scale = scale;
	c.conj = conj;
	c.fmac = fmac;

	for (int n = 0; n < 3; n++) {

		c.shifts[n] = 2. * M_PI * (float)(shifts[n]) / ((float)img_dims[n]);
		c.cn -= c.shifts[n] * (float)img_dims[n] / 2.;
		
		c.dims[n] = img_dims[n];
		c.tot *= c.dims[n];
	}

	c.N = md_calc_size(N - 3, img_dims + 3);

	const void* func = (const void*)kern_apply_linphases_3D;
	kern_apply_linphases_3D<<<getGridSize3(c.dims, func), getBlockSize3(c.dims, (const void*)func), 0, cuda_get_stream()>>>(c, (hipFloatComplex*)dst, (const hipFloatComplex*)src);
}


static __device__ double ftkb(double beta, double x)
{
	double a = pow(beta, 2.) - pow(M_PI * x, 2.);

	if (0. == a)
		return 1;

	if (a > 0)
		return (sinh(sqrt(a)) / sqrt(a));
	else
		return (sin(sqrt(-a)) / sqrt(-a));
}

static __device__ double rolloff(double x, double beta, double width)
{
	return 1. / ftkb(beta, x * width) / width;
}

static __device__ float posf(int d, int i)
{
	return (1 == d) ? 0. : (((float)i - (float)d / 2.) / (float)d);
}

struct rolloff_conf {

	long dims[3];
	long tot;
	long N;
	float os;
	float width;
	float beta;
	float bessel_beta;
};

__global__ void kern_apply_rolloff_correction(struct rolloff_conf c, hipFloatComplex* dst, const hipFloatComplex* src)
{
	int startX = threadIdx.x + blockDim.x * blockIdx.x;
	int strideX = blockDim.x * gridDim.x;

	int startY = threadIdx.y + blockDim.y * blockIdx.y;
	int strideY = blockDim.y * gridDim.y;

	int startZ = threadIdx.z + blockDim.z * blockIdx.z;
	int strideZ = blockDim.z * gridDim.z;

	for (long z = startZ; z < c.dims[2]; z += strideZ)
		for (long y = startY; y < c.dims[1]; y += strideY)
			for (long x = startX; x < c.dims[0]; x +=strideX) {

				long idx = x + c.dims[0] * (y + c.dims[1] * z);
				
				float val = ((c.dims[0] > 1) ? rolloff(posf(c.dims[0], x) / c.os, c.beta, c.width) * c.bessel_beta : 1)
					  * ((c.dims[1] > 1) ? rolloff(posf(c.dims[1], y) / c.os, c.beta, c.width) * c.bessel_beta : 1)
					  * ((c.dims[2] > 1) ? rolloff(posf(c.dims[2], z) / c.os, c.beta, c.width) * c.bessel_beta : 1);

				for (long i = 0; i < c.N; i++) {

					dst[idx + i * c.tot].x = val * src[idx + i * c.tot].x;
					dst[idx + i * c.tot].y = val * src[idx + i * c.tot].y;
				}
			}
}



extern "C" void cuda_apply_rolloff_correction(float os, float width, float beta, int N, const long dims[], _Complex float* dst, const _Complex float* src)
{
	struct rolloff_conf c = {

		.dims = { dims[0], dims[1], dims[2] },
		.tot = md_calc_size(3, dims),
		.N = md_calc_size(N - 3, dims + 3),
		.os = os,
		.width = width,
		.beta = beta,
		.bessel_beta = bessel_kb_beta,
	};

	const void* func = (const void*)kern_apply_rolloff_correction;
	kern_apply_rolloff_correction<<<getGridSize3(c.dims, func), getBlockSize3(c.dims, (const void*)func), 0, cuda_get_stream()>>>(c, (hipFloatComplex*)dst, (const hipFloatComplex*)src);
}


// Linear interpolation
__device__ static __inline__ float lerp(float a, float b, float c)
{
	return (1. - c) * a + c * b;
}

// Linear interpolation look up
__device__ static float intlookup(int n, const float* table, float x)
{
	x *= 2;
	int index = (int)(x * (n - 1));
	float fpart = x * (n - 1) - (float)index;
	float l = lerp(table[index], table[index + 1], fpart);
	return l;
}


static const struct multiplace_array_s* kb_table = NULL;

static void kb_precompute_gpu(double beta)
{
	#pragma omp critical(kb_tbale_gpu)
	if (NULL == kb_table)
		kb_table = kb_get_table(beta);
}

#define GRID_DIMS 3

struct grid_data {

	float os;
	float width;
	bool periodic;

	long samples;
	long grid_dims[4];
	long ksp_dims[4];

	float shift[GRID_DIMS];

	int ch;
	long off_ch_ksp;
	long off_ch_grid;

	int kb_size;
	float* kb_table;

	long NB;
	long SB_trj;
	long SB_grd;
	long SB_ksp;
};

struct grid_data_device {

	float pos[GRID_DIMS];
	int pos_grid[GRID_DIMS];
	int sti[GRID_DIMS];
	int eni[GRID_DIMS];
	int off[GRID_DIMS];
};

__device__ static __inline__ void dev_atomic_zadd_scl(hipFloatComplex* arg, hipFloatComplex val, float scl)
{
	atomicAdd(&(arg->x), val.x * scl);
	atomicAdd(&(arg->y), val.y * scl);
}

#if 0
__device__ static __inline__ void dev_zadd_scl(hipFloatComplex* arg, hipFloatComplex val, float scl)
{
	arg->x += val.x * scl;
	arg->y += val.y * scl;
}
#endif

__device__ static void grid_point_r(const struct grid_data* gd, const struct grid_data_device* gdd, hipFloatComplex* dst, const hipFloatComplex* src)
{

	float d[GRID_DIMS];
	long ind[GRID_DIMS];

	for (long z = gdd->sti[2]; z <= gdd->eni[2]; z++) {

		d[2] = intlookup(gd->kb_size, gd->kb_table, fabs(((float)z - gdd->pos[2]))/ gd->width);
		ind[2] = ((z + gdd->off[2]) % gd->grid_dims[2]);

		for (long y = gdd->sti[1]; y <= gdd->eni[1]; y++) {

			d[1] = intlookup(gd->kb_size, gd->kb_table, fabs(((float)y - gdd->pos[1]))/ gd->width) * d[2];
			ind[1] = ((y + gdd->off[1]) % gd->grid_dims[1]) + ind[2] * gd->grid_dims[1];

			for (long x = gdd->sti[0]; x <= gdd->eni[0]; x++) {

				d[0] = intlookup(gd->kb_size, gd->kb_table, fabs(((float)x - gdd->pos[0]))/ gd->width) * d[1];
				ind[0] = ((x + gdd->off[0]) % gd->grid_dims[0]) + ind[1] * gd->grid_dims[0];

				dev_atomic_zadd_scl(dst + ind[0] , src[0], d[0]);
			}
		}
	}
}

__device__ static void gridH_point_r(const struct grid_data* gd, const struct grid_data_device* gdd, hipFloatComplex* dst, const hipFloatComplex* src)
{

	float d[GRID_DIMS];
	long ind[GRID_DIMS];

	for (long z = gdd->sti[2]; z <= gdd->eni[2]; z++) {

		d[2] = intlookup(gd->kb_size, gd->kb_table, fabs(((float)z - gdd->pos[2]))/ gd->width);
		ind[2] = ((z + gdd->off[2]) % gd->grid_dims[2]);

		for (long y = gdd->sti[1]; y <= gdd->eni[1]; y++) {

			d[1] = intlookup(gd->kb_size, gd->kb_table, fabs(((float)y - gdd->pos[1]))/ gd->width) * d[2];
			ind[1] = ((y + gdd->off[1]) % gd->grid_dims[1]) + ind[2] * gd->grid_dims[1];

			for (long x = gdd->sti[0]; x <= gdd->eni[0]; x++) {

				d[0] = intlookup(gd->kb_size, gd->kb_table, fabs(((float)x - gdd->pos[0]))/ gd->width) * d[1];
				ind[0] = ((x + gdd->off[0]) % gd->grid_dims[0]) + ind[1] * gd->grid_dims[0];

				dev_atomic_zadd_scl(dst, src[ind[0]], d[0]);
			}
		}
	}
}

__device__ static struct grid_data_device get_grid_data_device(const struct grid_data* conf, const hipFloatComplex traj[GRID_DIMS])
{
	struct grid_data_device gdd;

	for (int j = 0; j < GRID_DIMS; j++) {

		gdd.pos[j] = conf->os * ((traj[j]).x + conf->shift[j]);
		gdd.pos[j] += (conf->grid_dims[j] > 1) ? ((float) conf->grid_dims[j] / 2.) : 0.;

		gdd.sti[j] = (int)ceil(gdd.pos[j] - 0.5 * conf->width);
		gdd.eni[j] = (int)floor(gdd.pos[j] + 0.5 * conf->width);
		gdd.off[j] = 0;

		if (gdd.sti[j] > gdd.eni[j])
			continue;

		if (!conf->periodic) {

			gdd.sti[j] = MAX(gdd.sti[j], 0);
			gdd.eni[j] = MIN(gdd.eni[j], conf->grid_dims[j] - 1);

		} else {

			while (gdd.sti[j] + gdd.off[j] < 0)
				gdd.off[j] += conf->grid_dims[j];
		}

		if (1 == conf->grid_dims[j]) {

			assert(0. == gdd.pos[j]); // ==0. fails nondeterministically for test_nufft_forward bbdec08cb
			gdd.sti[j] = 0;
			gdd.eni[j] = 0;
		}
	}

	return gdd;
}


__global__ static void kern_grid(struct grid_data conf, const hipFloatComplex* traj, hipFloatComplex* grid, const hipFloatComplex* ksp)
{
	int start[3];
	int stride[3];

	start[0] = threadIdx.x + blockDim.x * blockIdx.x;
	start[1] = threadIdx.y + blockDim.y * blockIdx.y;
	start[2] = threadIdx.z + blockDim.z * blockIdx.z;

	stride[0] = blockDim.x * gridDim.x;
	stride[1] = blockDim.y * gridDim.y;
	stride[2] = blockDim.z * gridDim.z;


	struct grid_data_device gdd;

	for (long z = start[2]; z < conf.NB; z +=stride[2])
	for (long c = start[1]; c < conf.ch; c += stride[1])
	for (long i = start[0]; i < conf.samples; i += stride[0]) {

		long offset_trj = z * conf.SB_trj;
		long offset_grd = z * conf.SB_grd + c * conf.off_ch_grid;
		long offset_ksp = z * conf.SB_ksp + c * conf.off_ch_ksp;

		gdd = get_grid_data_device(&conf, traj + offset_trj + i * GRID_DIMS);

		grid_point_r(&conf, &gdd, grid + offset_grd, ksp + i + offset_ksp);
	}
}


void cuda_grid(const struct grid_conf_s* conf, int N, const long traj_dims[], const _Complex float* traj, const long grid_dims[], _Complex float* grid, const long ksp_dims[], const _Complex float* src)
{

	kb_precompute_gpu(conf->beta);

	assert((4 == N) || (5 == N));

	struct grid_data gd = {

		.os = conf->os,
		.width = conf->width,
		.periodic = conf->periodic,

		.samples = ksp_dims[1] * ksp_dims[2],

		.grid_dims = { grid_dims[0], grid_dims[1], grid_dims[2], grid_dims[3]},
		.ksp_dims = { ksp_dims[0], ksp_dims[1], ksp_dims[2], ksp_dims[3]},

		.shift = { conf->shift[0], conf->shift[1], conf->shift[2] },

		.ch = (int)ksp_dims[3],

		.off_ch_ksp = md_calc_size(3, ksp_dims),
		.off_ch_grid = md_calc_size(3, grid_dims),

		.kb_size = kb_size,
		.kb_table = (float*)multiplace_read((struct multiplace_array_s*)kb_table, (const void*)traj),

		.NB = (4 == N) ? 1 : MAX(ksp_dims[4], grid_dims[4]),
		.SB_trj = ((4 == N) || (1 == traj_dims[4])) ? 0 : md_calc_size(4, traj_dims),
		.SB_grd = ((4 == N) || (1 == grid_dims[4])) ? 0 : md_calc_size(4, grid_dims),
		.SB_ksp = ((4 == N) || (1 == ksp_dims[4])) ? 0 : md_calc_size(4, ksp_dims),
	};

	const long size[3] = { gd.samples, gd.ch, gd.NB };
	dim3 cu_block = getBlockSize3(size, (const void*)kern_grid);
	dim3 cu_grid = getGridSize3(size, (const void*)kern_grid);

	kern_grid<<<cu_grid, cu_block, 0, cuda_get_stream() >>>(gd, (const hipFloatComplex*)traj, (hipFloatComplex*)grid, (const hipFloatComplex*)src);

	CUDA_KERNEL_ERROR;
}

__global__ static void kern_gridH(struct grid_data conf, const hipFloatComplex* traj, hipFloatComplex* ksp, const hipFloatComplex* grid)
{
	int start[3];
	int stride[3];

	start[0] = threadIdx.x + blockDim.x * blockIdx.x;
	start[1] = threadIdx.y + blockDim.y * blockIdx.y;
	start[2] = threadIdx.z + blockDim.z * blockIdx.z;

	stride[0] = blockDim.x * gridDim.x;
	stride[1] = blockDim.y * gridDim.y;
	stride[2] = blockDim.z * gridDim.z;


	struct grid_data_device gdd;

	for (long z = start[2]; z < conf.NB; z +=stride[2])
	for (long c = start[1]; c < conf.ch; c += stride[1])
	for (long i = start[0]; i < conf.samples; i += stride[0]) {

		long offset_trj = z * conf.SB_trj;
		long offset_grd = z * conf.SB_grd + c * conf.off_ch_grid;
		long offset_ksp = z * conf.SB_ksp + c * conf.off_ch_ksp;

		gdd = get_grid_data_device(&conf, traj + offset_trj + i * GRID_DIMS);

		gridH_point_r(&conf, &gdd, ksp + i + offset_ksp, grid + offset_grd);
	}
}


void cuda_gridH(const struct grid_conf_s* conf, int N, const long traj_dims[], const _Complex float* traj, const long ksp_dims[], _Complex float* dst, const long grid_dims[], const _Complex float* grid)
{

	kb_precompute_gpu(conf->beta);

	assert((4 == N) || (5 == N));

	struct grid_data gd = {

		.os = conf->os,
		.width = conf->width,
		.periodic = conf->periodic,

		.samples = ksp_dims[1] * ksp_dims[2],

		.grid_dims = { grid_dims[0], grid_dims[1], grid_dims[2], grid_dims[3]},
		.ksp_dims = { ksp_dims[0], ksp_dims[1], ksp_dims[2], ksp_dims[3]},

		.shift = { conf->shift[0], conf->shift[1], conf->shift[2] },

		.ch = (int)ksp_dims[3],

		.off_ch_ksp = md_calc_size(3, ksp_dims),
		.off_ch_grid = md_calc_size(3, grid_dims),

		.kb_size = kb_size,
		.kb_table = (float*)multiplace_read((struct multiplace_array_s*)kb_table, (const void*)traj),

		.NB = (4 == N) ? 1 : MAX(ksp_dims[4], grid_dims[4]),
		.SB_trj = ((4 == N) || (1 == traj_dims[4])) ? 0 : md_calc_size(4, traj_dims),
		.SB_grd = ((4 == N) || (1 == grid_dims[4])) ? 0 : md_calc_size(4, grid_dims),
		.SB_ksp = ((4 == N) || (1 == ksp_dims[4])) ? 0 : md_calc_size(4, ksp_dims),
	};

	const long size[3] = { gd.samples, gd.ch, gd.NB };
	dim3 cu_block = getBlockSize3(size, (const void*)kern_gridH);
	dim3 cu_grid = getGridSize3(size, (const void*)kern_gridH);

	kern_gridH<<<cu_grid, cu_block, 0, cuda_get_stream() >>>(gd, (const hipFloatComplex*)traj, (hipFloatComplex*)dst, (const hipFloatComplex*)grid);

	CUDA_KERNEL_ERROR;
}

